#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ruby.h>
#include <hip/hip_runtime.h>

__global__ 
void my_kernel()
{
  printf("Hello from thread %d block %d\n", threadIdx.x, blockIdx.x);
}

VALUE 
print_from_kernel(VALUE obj)
{
  my_kernel<<<2,2>>>();
  hipDeviceSynchronize();
  return Qnil;
}

VALUE 
printGPUinfo(VALUE obj)
{
  int devID, count = 0;
  hipDeviceProp_t props;
  
  hipGetDeviceCount(&count);
  if(count == 0){
    printf("CUDA Device Not Found.\n");
    return Qnil;
  }
  for(devID = 0; devID < count; devID++){
    if(hipGetDevice(&devID) == hipSuccess && hipGetDeviceProperties(&props, devID) == hipSuccess){
      printf("GPU %d: \"%s\" with Compute %d.%d capability\n",
             devID, props.name, props.major, props.minor);
    }else{
      printf("Getting CUDA Device %d info failed.\n", devID);
    }
  }
  hipDeviceReset();
  return Qnil;
}

extern "C" void
Init_culib(){
  VALUE mCulib = rb_define_module("Culib");
  rb_define_singleton_method(mCulib, "print_from_kernel", RUBY_METHOD_FUNC(print_from_kernel), 0);
  rb_define_singleton_method(mCulib, "printGPUinfo", RUBY_METHOD_FUNC(printGPUinfo), 0);
}

